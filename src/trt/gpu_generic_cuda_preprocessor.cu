#include "hip/hip_runtime.h"
/**
 * @file gpu_generic_cuda_preprocessor.cu
 * @author Sinter Wong (sintercver@gmail.com)
 * @brief
 * @version 0.1
 * @date 2025-07-14
 *
 * @copyright Copyright (c) 2025
 *
 */
#include "ai_core/typed_buffer.hpp"
#include "cuda_utils.hpp"
#include "gpu_generic_cuda_preprocessor.hpp"
#include "trt_device_buffer.hpp"
#include "trt_utils.hpp"

#include <logger.hpp>
#include <opencv2/core.hpp>

namespace ai_core::dnn::gpu
{

  TypedBuffer GpuGenericCudaPreprocessor::process(const FramePreprocessArg &args,
                                                  const FrameInput &input,
                                                  FrameTransformContext &runtimeArgs) const
  {
    if (input.image == nullptr)
    {
      LOG_ERRORS << "Input frame is null.";
      throw std::runtime_error("Input frame is null.");
    }

    if (input.inputRoi == nullptr)
    {
      runtimeArgs.roi = std::make_shared<cv::Rect>(0, 0, input.image->cols,
                                                   input.image->rows);
    }
    else
    {
      runtimeArgs.roi = input.inputRoi;
    }
    runtimeArgs.originShape = {input.image->cols, input.image->rows,
                               input.image->channels()};

    const auto &image = *input.image;
    const auto &roi = *runtimeArgs.roi;
    if (roi.x < 0 || roi.y < 0 || roi.width <= 0 || roi.height <= 0 ||
        roi.x + roi.width > image.cols || roi.y + roi.height > image.rows)
    {
      LOG_ERRORS << "Invalid ROI: " << roi << " for image size: " << image.size();
      throw std::runtime_error("Invalid ROI.");
    }

    const uint8_t *pSrcData = image.data;
    int src_h = image.rows;
    int src_w = image.cols;
    int src_c = image.channels();

    if (roi.area() > 0)
    {
      src_h = roi.height;
      src_w = roi.width;
    }

    trt_utils::TrtDeviceBuffer d_inputImage(image.total() * image.elemSize());
    CHECK_CUDA(hipMemcpy(d_inputImage.get(), pSrcData,
                          image.total() * image.elemSize(),
                          hipMemcpyHostToDevice));

    trt_utils::TrtDeviceBuffer d_mean(args.meanVals.size() * sizeof(float));
    trt_utils::TrtDeviceBuffer d_std(args.normVals.size() * sizeof(float));
    CHECK_CUDA(hipMemcpy(d_mean.get(), args.meanVals.data(),
                          args.meanVals.size() * sizeof(float),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_std.get(), args.normVals.data(),
                          args.normVals.size() * sizeof(float),
                          hipMemcpyHostToDevice));

    size_t totalElements = (size_t)args.modelInputShape.c *
                           args.modelInputShape.h * args.modelInputShape.w;
    size_t byteSizeFP32 = totalElements * sizeof(float);
    TypedBuffer hwcBuffer =
        TypedBuffer::createFromGpu(DataType::FLOAT32, byteSizeFP32);

    if (args.isEqualScale)
    {
      float scale = std::min(static_cast<float>(args.modelInputShape.w) / src_w,
                             static_cast<float>(args.modelInputShape.h) / src_h);
      int new_w = static_cast<int>(src_w * scale);
      int new_h = static_cast<int>(src_h * scale);

      runtimeArgs.leftPad = (args.modelInputShape.w - new_w) / 2;
      runtimeArgs.topPad = (args.modelInputShape.h - new_h) / 2;

      trt_utils::TrtDeviceBuffer d_pad(args.pad.size() * sizeof(float));
      CHECK_CUDA(hipMemcpy(d_pad.get(), args.pad.data(),
                            args.pad.size() * sizeof(float),
                            hipMemcpyHostToDevice));

      const uint8_t *d_roiImage_ptr =
          (const uint8_t *)d_inputImage.get() +
          ((size_t)roi.y * image.cols + roi.x) * src_c;

      cuda_op::escale_resize_normalize_gpu(
          d_roiImage_ptr, (float *)hwcBuffer.getRawDevicePtr(), src_h, src_w,
          src_c, args.modelInputShape.h, args.modelInputShape.w,
          (const float *)d_mean.get(), (const float *)d_std.get(),
          (const float *)d_pad.get());
    }
    else
    {
      cuda_op::crop_resize_normalize_gpu(
          (const uint8_t *)d_inputImage.get(),
          (float *)hwcBuffer.getRawDevicePtr(), image.rows, image.cols, src_c,
          roi.x, roi.y, src_h, src_w, args.modelInputShape.h,
          args.modelInputShape.w, (const float *)d_mean.get(),
          (const float *)d_std.get());
    }

    size_t finalByteSize =
        totalElements * TypedBuffer::getElementSize(args.dataType);
    TypedBuffer finalDeviceBuffer =
        TypedBuffer::createFromGpu(args.dataType, finalByteSize);

    if (args.hwc2chw)
    {
      TypedBuffer chwBuffer =
          TypedBuffer::createFromGpu(DataType::FLOAT32, byteSizeFP32);
      cuda_op::hwc_to_chw_gpu((const float *)hwcBuffer.getRawDevicePtr(),
                              (float *)chwBuffer.getRawDevicePtr(),
                              args.modelInputShape.h, args.modelInputShape.w,
                              args.modelInputShape.c);

      if (args.dataType == DataType::FLOAT16)
      {
        cuda_op::fp32_to_fp16_gpu((const float *)chwBuffer.getRawDevicePtr(),
                                  (uint16_t *)finalDeviceBuffer.getRawDevicePtr(),
                                  totalElements);
      }
      else
      {
        CHECK_CUDA(hipMemcpy(finalDeviceBuffer.getRawDevicePtr(),
                              chwBuffer.getRawDevicePtr(), finalByteSize,
                              hipMemcpyDeviceToDevice));
      }
    }
    else
    {
      if (args.dataType == DataType::FLOAT16)
      {
        cuda_op::fp32_to_fp16_gpu((const float *)hwcBuffer.getRawDevicePtr(),
                                  (uint16_t *)finalDeviceBuffer.getRawDevicePtr(),
                                  totalElements);
      }
      else
      {
        CHECK_CUDA(hipMemcpy(finalDeviceBuffer.getRawDevicePtr(),
                              hwcBuffer.getRawDevicePtr(), finalByteSize,
                              hipMemcpyDeviceToDevice));
      }
    }

    if (args.outputLocation == BufferLocation::GPU_DEVICE)
    {
      return finalDeviceBuffer;
    }
    else
    {
      std::vector<uint8_t> hostData(finalByteSize);
      CHECK_CUDA(hipMemcpy(hostData.data(), finalDeviceBuffer.getRawDevicePtr(),
                            finalByteSize, hipMemcpyDeviceToHost));
      return TypedBuffer::createFromCpu(args.dataType, std::move(hostData));
    }
  }

  TypedBuffer GpuGenericCudaPreprocessor::batchProcess(const FramePreprocessArg &args,
                                                       const std::vector<FrameInput> &input,
                                                       std::vector<FrameTransformContext> &runtimeArgs) const
  {
    LOG_ERRORS << "Batch processing not implemented for GpuGenericCudaPreprocessor";
    return TypedBuffer();
  }

} // namespace ai_core::dnn::gpu
